#include "hip/hip_runtime.h"
// %%writefile cuda_filtro.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <unistd.h>
#include <pthread.h>
#include <sys/time.h>

#include "sod/sod.h"

#define MAX_H 4320
#define MAX_W 8192

#include <hip/hip_runtime.h>

char* IMAGEN_ENTRADA;
char* IMAGEN_SALIDA;
int ARG;
int NUM_HILOS;
int BLOQUES_GPU;
int HILOS_GPU;

#define MAX_INTERVAL 2048
int intervalo[MAX_INTERVAL][2]; // El i-th hilo vá desde el intervalo intervalo[i][0] hasta intervalo[i][1]

// ref: https://kth.instructure.com/courses/12406/pages/timing-your-kernel-cpu-timer-and-nvprof?module_item_id=169241
double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

#define INIT_KERNEL float kernel[3][3] = {{-1,-1,-1}, {-1, 8,-1}, {-1,-1,-1}};

__global__ 
void kernel(int d_intervalo[MAX_INTERVAL][2], float (*d_board)[MAX_W], float (*d_output)[MAX_W], int *d_W, int *d_blocks) {

    int ID = blockIdx.x * blockDim.x + threadIdx.x;

    if (ID < (*d_blocks)) {
        
        INIT_KERNEL;

        // Obtener intervalo, desde-hasta
        int from = d_intervalo[ID][0];
        int to = d_intervalo[ID][1];

        for(int y = from; y <= to; ++y) {
            for(int x = 1; x < (*d_W)-1; ++x) {

                float sum = 0.0;
                // Iterar la matrix 3x3 kernel
                for(int ky = -1; ky <= 1; ++ky) {
                    for(int kx = -1; kx <= 1; ++kx) {
                        // Obtener pixel (Red) en la coordenada (x+kx, y+ky)
                        float val = d_board[x+kx][y+ky]; // R
                        sum += kernel[ky+1][kx+1] * val;

                        //  kernel       pixeles de la imagen, px=Pixel

                        // [k1 k2 k3]    [px1 px2 px3]
                        // [k4 k5 k6]    [px4 px5 px6]
                        // [k7 k8 k9]    [px7 px8 px9]

                        // sum = k1*px1 + k2*px2 + ... + k9*px9
                    }
                }
                d_output[x][y] = abs(sum);
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if(argc < 6) {
        printf("Debe proporcionar 5 argumentos: [imagen de entrada] [imagen de salida] [argumento del filtro] [numero de bloques] [numero de hilos por bloque]");
        // Ejemplo: ./filtro.o img/input1.png img/output1.png 8 16
        exit(0);
    }
    // Ruta de la imagen de entrada: Ej: img/input1.png
    IMAGEN_ENTRADA = argv[1];

    // Ruta de la imagen de salida: Ej: img/output1.png
    IMAGEN_SALIDA = argv[2];

    // Argumento del filtro: Ej 8
    ARG = atof(argv[3]);
    // kernel[1][1] = ARG;

    sod_img imgIn;
    sod_img imgOut;

    // w=4096 h=2160

    BLOQUES_GPU = atoi(argv[4]);

    HILOS_GPU = atoi(argv[5]);

    NUM_HILOS = BLOQUES_GPU * HILOS_GPU;

    // Cargar Imagen en memoria
    imgIn = sod_img_load_from_file(IMAGEN_ENTRADA, SOD_IMG_COLOR);
    imgOut = sod_img_load_from_file(IMAGEN_ENTRADA, SOD_IMG_COLOR);

    if (imgIn.data == 0) {
        // Validar que la imagen exista
        printf("No pudo cargar la imagen %s\n", IMAGEN_ENTRADA);
        return 0;
    }

    // Definir intervalos para NUM_HILOS hilos
    int factor = imgIn.h / NUM_HILOS;
    int last = 1;
    for(int i = 0; i < NUM_HILOS; ++i) {
        intervalo[i][0] = last;
        if(i != (NUM_HILOS-1)) {
            intervalo[i][1] = last + factor-1;
        } else {
            intervalo[i][1] = imgIn.h - 1;
        }
        last = intervalo[i][1] + 1;
    }

    printf("w=%d h=%d\n", imgIn.w, imgIn.h);

    // Crear variables con memoria dinamica
    // host board
    float (*board)[MAX_W] = (float (*)[MAX_W]) malloc(MAX_H*MAX_W*sizeof(float));

    for(int y = 0; y <= imgIn.h; ++y) {
        for(int x = 0; x < imgIn.w; ++x) {
            float val = sod_img_get_pixel(imgIn, x, y, 0); // RED
            board[x][y] = val;
        }
    }

    // device board
    float (*d_board)[MAX_W];
    hipMalloc(&d_board, MAX_H*MAX_W*sizeof(float));
    hipMemcpy(d_board, board, MAX_H*MAX_W*sizeof(float), hipMemcpyHostToDevice);

    // Device output
    float (*d_output)[MAX_W];
    hipMalloc(&d_output, MAX_H*MAX_W*sizeof(float));

    // Device Intervalo
    int (*d_intervalo)[2];
    hipMalloc(&d_intervalo, MAX_INTERVAL*2*sizeof(int));
    hipMemcpy(d_intervalo, intervalo, MAX_INTERVAL*2*sizeof(int), hipMemcpyHostToDevice);

    // Device W
    int *d_W;
    int *tmp_W;
    int tmp = imgIn.w;
    tmp_W = &tmp;
    hipMalloc(&d_W, sizeof(int));
    hipMemcpy(d_W, tmp_W, sizeof(int), hipMemcpyHostToDevice);

    int *d_blocks;
    int *hilos;
    hilos = &NUM_HILOS;
    hipMalloc(&d_blocks, sizeof(int));
    hipMemcpy(d_blocks, hilos, sizeof(int), hipMemcpyHostToDevice);

    // Medir Tiempo de Ejecución
    double start = cpuSecond();

    // Run Cuda
    kernel<<<BLOQUES_GPU, HILOS_GPU>>>(d_intervalo, d_board, d_output, d_W, d_blocks);

    hipDeviceSynchronize();
    double stop = cpuSecond();

    hipMemcpy(board, d_output, MAX_H*MAX_W*sizeof(float), hipMemcpyDeviceToHost);

    for(int y = 0; y <= imgIn.h; ++y) {
        for(int x = 0; x < imgIn.w; ++x) {
            float val = board[x][y];
            sod_img_set_pixel(imgOut, x, y, 0, abs(val)); // R
            sod_img_set_pixel(imgOut, x, y, 1, abs(val)); // G
            sod_img_set_pixel(imgOut, x, y, 2, abs(val)); // B
        }
    }
    
    // Guardar la Imagen
    sod_img_save_as_png(imgOut, IMAGEN_SALIDA);

    // Liberar la Memoria
    sod_free_image(imgIn);
    sod_free_image(imgOut);

    double time_elapsed = stop - start;

    // Mostrar el tiempo de ejecución
    printf("\nTime elapsed: %.8f sec using blocks_gpu=%d, threads_per_blocks=%d\n", time_elapsed, BLOQUES_GPU, HILOS_GPU);
    fflush(stdout);

    return 0;
}